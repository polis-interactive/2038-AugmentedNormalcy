
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    printf("hello from the cpu \n");
    cuda_hello<<<1,1>>>();
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
    return 0;
}

